
#include <hip/hip_runtime.h>
#include <stdio.h>
#define DEBUGPRINT 0
__global__ void compute_grid_h_kernel( double *gridh, double *xm1, double *ym1, double *zm1, int nelt, int lx1, int ly1, int lz1, int if3d,  int nnel,int lxy, int lxyz  ){    
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<nnel){

		int ix= id % lx1;
		int iy= (id/lx1)%ly1;
		int iz = (id / (lxy))%lz1;
		int e =  id / (lxyz);
		int km1,kp1,izm,izp;
                int x1,x2,x3,x4,x5,x6,y1,y2,y3,y4,y5,y6,z1,z2,z3,z4,z5,z6;
                double a1,a2,a3,b1,b2,b3,c1,c2,c3,d1,d2,d3;
		if (if3d) {
			km1=iz-1;
			kp1=iz+1;
			izm=km1;
			if (km1 < 1){ izm=iz;}
			izp=kp1;
			if (kp1 > lz1) {izp=iz;}
		}
		else {
			izm=iz;
			izp=iz;
		}

	        int jm1=iy-1;
		int jp1=iy+1;
		int iym=jm1;
		if (jm1 < 1) {iym=iy;}
		int iyp=jp1;
		if (jp1 > ly1) {iyp=iy;}

		int im1=ix-1;
		int ip1=ix+1;
		int ixm=im1;
		if (im1 < 1){ ixm=ix;}
		int ixp=ip1;
		if (ip1 > lx1) {ixp=ix;}
		x1 = xm1[e*lxyz+iz*lxy+iy*lx1+ixm];// possible  optimizations for array reads . adeesha
		x2 = xm1[e*lxyz+iz*lxy+iy*lx1+ixp];
		x3 = xm1[e*lxyz+iz*lxy+iym*lx1+ix];
		x4 = xm1[e*lxyz+iz*lxy+iyp*lx1+ix];
		x5 = xm1[e*lxyz+izm*lxy+iy*lx1+ix];
		x6 = xm1[e*lxyz+izp*lxy+iy*lx1+ix];
		y1 = ym1[e*lxyz+iz*lxy+iy*lx1+ixm];
		y2 = ym1[e*lxyz+iz*lxy+iy*lx1+ixp];
		y3 = ym1[e*lxyz+iz*lxy+iym*lx1+ix];
		y4 = ym1[e*lxyz+iz*lxy+iyp*lx1+ix];
		y5 = ym1[e*lxyz+izm*lxy+iy*lx1+ix];
		y6 = ym1[e*lxyz+izp*lxy+iy*lx1+ix];
		z1 = zm1[e*lxyz+iz*lxy+iy*lx1+ixm];
		z2 = zm1[e*lxyz+iz*lxy+iy*lx1+ixp];
		z3 = zm1[e*lxyz+iz*lxy+iym*lx1+ix];
		z4 = zm1[e*lxyz+iz*lxy+iyp*lx1+ix];
		z5 = zm1[e*lxyz+izm*lxy+iy*lx1+ix];
		z6 = zm1[e*lxyz+izp*lxy+iy*lx1+ix];
		a1=x2-x1;
		a2=y2-y1;
		a3=z2-z1;
		b1=x4-x3;
		b2=y4-y3;
		b3=z4-z3;
		c1=x6-x5;
		c2=y6-y5;
		c3=z6-z5;
		double fact;
		if (if3d) {
			fact=0.125; // h doesn't reach into corners of neighboring elements
			if (ixp==ix || ixm==ix){ fact=2.0*fact;}
			if (iym==iy || iyp==iy) {fact=2.0*fact;}
			if (izm==iz||izp==iz) {fact=2.0*fact;}
			//call cross(d,a,b);
			// cartesian vector cross product in gpu
                        d1 = a2*b3 - a3*b2;
      			d2 = a3*b1 - a1*b3;
      			d3 = a1*b2 - a2*b1;
                        // vector dot product to get sum
                        double sum = c1*d1+c2*d2+c3*d3;
			gridh[e*lxyz+iz*lxy+iy*lx1+ix]=fact*sum;
			gridh[e*lxyz+iz*lxy+iy*lx1+ix]=pow(fabs(gridh[e*lxyz+iz*lxy+iy*lx1+ix]),(1.0/3.0));
		}
		else{
			fact=0.25;
			if (ixp==ix||ixm==ix) fact=2.0*fact;
			if (iym==iy||iyp==iy) fact=2.0*fact;
			gridh[e*lxyz+iz*lxy+iy*lx1+ix]=sqrtf(fact*fabs(a1*b2-a2*b1));
		}





	}
}


extern "C" void compute_grid_h_gpu_wrapper_(int *glbblockSize1,double *d_gridh, double *d_xm1, double *d_ym1, double *d_zm1, int *nelt, int *lx1, int *ly1, int *lz1, int *if3d){
#ifdef DEBUGPRINT
hipDeviceSynchronize();
 hipError_t code1 = hipPeekAtLastError();

printf("CUDA: Start compute_grid_h_gpu_wrapper cuda status: %s\n",hipGetErrorString(code1));

printf("CUDA: Start compute_grid_h_gpu_wrapper values glbblockSize1=%d, nelt=%d,lx1=%d,ly1=%d,lz1=%d,if3d=%d\n",glbblockSize1[0],nelt[0],lx1[0],ly1[0],lz1[0],if3d[0]);
#endif
	int blockSize = glbblockSize1[0], gridSize;
	int lxy=lx1[0]*ly1[0];
	int lxyz=lxy*lz1[0];
	int nnel=nelt[0]*lxyz;
	gridSize = (int)ceil((float)nnel/blockSize);
#ifdef DEBUGPRINT
	printf("CUDA:  compute_grid_h_gpu_wrapper grid size = %d, block size = %d \n",gridSize,blockSize);
#endif
	compute_grid_h_kernel<<<gridSize, blockSize>>>(d_gridh, d_xm1, d_ym1, d_zm1, nelt[0],lx1[0],ly1[0],lz1[0],if3d[0],nnel,lxy,lxyz);
#ifdef DEBUGPRINT
hipDeviceSynchronize();
 hipError_t code2 = hipPeekAtLastError();

printf("CUDA: End compute_grid_h_gpu_wrapper cuda status: %s\n",hipGetErrorString(code2));

#endif
 	/*printf(" $$$ compute_grid_h_gpu_wrapper check start ");
        for(int b=0;b<10;b++){
                printf("d_gridh[%d] = %lf \n",b,d_gridh[b]);
        }
        printf(" $$$ compute_grid_h_gpu_wrapper check End ");*/



}

__global__ void compute_mesh_h_kernel( double *meshh, double *xm1, double *ym1, double *zm1, int nelt, int lx1, int ly1, int lz1, int if3d, double rp , int ncrn, int lxy,  int lxyz){
        int id = blockIdx.x*blockDim.x+threadIdx.x;
        if(id<nelt){
                int ic1,ic2;
                //int ic2= id % ncrn;
                //int ic1= (id/ncrn)%ncrn;
                int e =  id;
                int km1,kp1,izm,izp;
               
		double xcrn[8],ycrn[8],zcrn[8];
			
               	int k1=1;
		int k2= lz1;
 		int j1=1;
		int j2= ly1;
		int i1=1;
		int i2= lx1;

		xcrn[0] = xm1[e*lxyz];
		xcrn[1] = xm1[e*lxyz+lx1-1];
		xcrn[2] = xm1[e*lxyz+(ly1-1)*lx1];
		xcrn[3] = xm1[e*lxyz+(ly1-1)*lx1+lx1-1];
		ycrn[0] = ym1[e*lxyz];
		ycrn[1] = ym1[e*lxyz+lx1-1];
		ycrn[2] = ym1[e*lxyz+(ly1-1)*lx1];
		ycrn[3]= ym1[e*lxyz+(ly1-1)*lx1+lx1-1];
		if (if3d) {
			xcrn[4] = xm1[e*lxyz+(lz1-1)*lxy];
			xcrn[5] = xm1[e*lxyz+(lz1-1)*lxy+lx1-1];
			xcrn[6] = xm1[e*lxyz+(lz1-1)*lxy+(ly1-1)*lx1];
			xcrn[7] = xm1[e*lxyz+(lz1-1)*lxy+(ly1-1)*lx1+lx1-1];
			ycrn[4] = ym1[e*lxyz+(lz1-1)*lxy];
			ycrn[5] = ym1[e*lxyz+(lz1-1)*lxy+lx1-1];
			ycrn[6] = ym1[e*lxyz+(lz1-1)*lxy+(ly1-1)*lx1];
			ycrn[7] = ym1[e*lxyz+(lz1-1)*lxy+(ly1-1)*lx1+lx1-1];
			zcrn[0] = zm1[e*lxyz];
			zcrn[1] = zm1[e*lxyz+lx1-1];
			zcrn[2] = zm1[e*lxyz+(ly1-1)*lx1];
			zcrn[3] = zm1[e*lxyz+(ly1-1)*lx1+lx1-1];
			zcrn[4] = zm1[e*lxyz+(lz1-1)*lxy];
			zcrn[5] = zm1[e*lxyz+(lz1-1)*lxy+lx1-1];
			zcrn[6] = zm1[e*lxyz+(lz1-1)*lxy+(ly1-1)*lx1];
			zcrn[7] = zm1[e*lxyz+(lz1-1)*lxy+(ly1-1)*lx1+lx1-1];
                }
                double dist = 10e+36;
                for (ic1 = 0 ; ic1<ncrn;ic1++){
			for (ic2 = 0; ic2<ncrn; ic2++){
				if(ic2 !=ic1){
                                	double txcrn = xcrn[ic2]-xcrn[ic1];
					double tycrn = ycrn[ic2]-ycrn[ic1];
					double tzcrn = zcrn[ic2]-zcrn[ic1];	
					double dtmp = txcrn*txcrn+tycrn*tycrn+tzcrn*tzcrn;
					double sqrtdtmp = sqrtf(dtmp) ;
					if(sqrtdtmp<dist){
						dist =sqrtdtmp;
					}
				}
			}
		}
		/*if(id==0){
			for(int aa=0;aa<2;aa++){
				printf("$$$ print from cuda xm1 = %lf, ym1 =%lf zm1 = %lf \n",xm1[aa],ym1[aa],zm1[aa]);
				printf("$$$ print from cuda xcrn = %lf, ycrn =%lf zcrn = %lf \n",xcrn[aa],ycrn[aa],zcrn[aa]);
			}
			printf("$$$ print from cuda dist = %lf, rp=%lf \n",dist,rp );
		}*/
		meshh[e]= dist*rp;



        }
}



extern "C" void compute_mesh_h_gpu_wrapper_(int *glbblockSize2,double *d_meshh, double *d_xm1, double *d_ym1, double *d_zm1, int *nelt, int *lx1, int *ly1, int *lz1, int *if3d, double *rp, int *ncrn){
#ifdef DEBUGPRINT
hipDeviceSynchronize();
 hipError_t code1 = hipPeekAtLastError();

printf("CUDA: Start compute_mesh_h_gpu_wrapper cuda status: %s\n",hipGetErrorString(code1));

printf("CUDA: Start compute_mesh_h_gpu_wrapper values nelt=%d,lx1=%d,ly1=%d,lz1=%d,if3d=%d rp=%lf,ncrn=%d \n",nelt[0],lx1[0],ly1[0],lz1[0],if3d[0],rp[0],ncrn[0]);
#endif

        int blockSize = glbblockSize2[0], gridSize;
        gridSize = (int)ceil((float)nelt[0]/blockSize);
	int lxy=lx1[0]*ly1[0];
	int lxyz=lz1[0]*lxy;
        compute_mesh_h_kernel<<<gridSize, blockSize>>>(d_meshh, d_xm1, d_ym1, d_zm1, nelt[0],lx1[0],ly1[0],lz1[0],if3d[0],rp[0],ncrn[0],lxy,lxyz);
#ifdef DEBUGPRINT
hipDeviceSynchronize();
hipError_t code2 = hipPeekAtLastError();

printf("CUDA: End compute_mesh_h_gpu_wrapper cuda status: %s\n",hipGetErrorString(code2));

#endif
	/*printf(" $$$ compute_mesh_h_gpu_wrapper check start ");
        for(int b=0;b<10;b++){
                printf("d_meshh[%d] = %lf \n",b,d_gridh[b]);
        }
        printf(" $$$ compute_mesh_h_gpu_wrapper check End ")*/

}

